#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixAdd(int *A, int *B, int *C, int size){
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        C[(ty * size) + (tx)] = A[(ty * size) + tx] + B[(ty * size) + tx];
}
int main(int argc, char **argv){
        int size, totalSize;
        int* matrixA = 0;
        int* matrixB = 0;
        int* matrixC = 0;
        printf("Matrix size: ");
        scanf("%d", &size);
        totalSize = size * size;
        int *local = (int*) malloc(totalSize * sizeof(int));
        hipMalloc((void**) &matrixA, sizeof(int) * totalSize);
        hipMalloc((void**) &matrixB, sizeof(int) * totalSize);
        hipMalloc((void**) &matrixC, sizeof(int) * totalSize);
        for(int i = 0; i < totalSize; i++){
                local[i] = rand() % 100;
        }
	hipMemcpy(matrixA, local, sizeof(int) * totalSize, hipMemcpyHostToDevice);
        hipMemcpy(matrixB, local, sizeof(int) * totalSize, hipMemcpyHostToDevice);
        dim3 dimBlock(size, size);
        dim3 dimGrid(1);
        matrixAdd<<<dimGrid, dimBlock>>>(matrixA, matrixB, matrixC, size);
        hipDeviceSynchronize();
        hipMemcpy(local, matrixC, sizeof(int) * totalSize, hipMemcpyDeviceToHost);
        for(int i = 0; i < totalSize; i++){
                if(i % size == 0){
                        printf("\n%d ", local[i]);
                }
                else{
                     	printf("%d ", local[i]);
                }
        }
	printf("\n");
}
