#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void arrayReduce(int *A, int *sum, int size){
        int thread = threadIdx.x;
        sum += A[thread];
}

int main(int argc, char **argv){
        int size;
        int* deviceArray = 0;
        int* sum = 0;
        int* localSum = (int*) malloc(4);
        printf("Array Size: ");
        scanf("%d", &size);
        int* local = (int*) malloc(size * sizeof(int));
        hipMalloc((void**) &deviceArray, sizeof(int) * size);
        hipMalloc((void**) &sum, 4);
        for(int i = 0; i < size; i++){
                local[i] = rand() % 100;
        }
    hipMemcpy(deviceArray, local, size * sizeof(int), hipMemcpyHostToDevice);
        dim3 dimBlock(size,1);
        arrayReduce<<<1,dimBlock>>>(deviceArray, sum, size);
        hipDeviceSynchronize();
        hipMemcpy(localSum, sum, sizeof(int), hipMemcpyDeviceToHost);
        printf("\nSum: %d\n", *localSum);
}

