#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixAdd(int *A, int *B, int *C, int size){
        int row = blockIdx.y;
        int col = threadIdx.x;
        C[(row * size) + col] = A[(row * size) + col] + B[(row * size) + col];
}

int main(int argc, char **argv){
        int size, totalSize;
        int* matrixA = 0;
        int* matrixB = 0;
        int* matrixC = 0;
        printf("Matrix size: ");
        scanf("%d", &size);
        totalSize = size * size;
        dim3 blockDim(size, 1);
        dim3 gridDim(1, size);
        int * local = (int*) malloc(totalSize * sizeof(int));
        hipMalloc((void**) &matrixA, sizeof(int) * totalSize);
        hipMalloc((void**) &matrixB, sizeof(int) * totalSize);
        hipMalloc((void**) &matrixC, sizeof(int) * totalSize);
        for(int i = 0; i < totalSize; i++){
                local[i] = rand() % 100;
        }
    hipMemcpy(matrixA, local, sizeof(int) * totalSize, hipMemcpyHostToDevice);
        hipMemcpy(matrixB, local, sizeof(int) * totalSize, hipMemcpyHostToDevice);
        matrixAdd<<<gridDim, blockDim>>>(matrixA, matrixB, matrixC, size);
        hipDeviceSynchronize();
        hipMemcpy(local, matrixC, sizeof(int) * totalSize, hipMemcpyDeviceToHost);
        for(int i = 0; i < totalSize; i++){
                if(i % size == 0){
                        printf("\n%d ", local[i]);
                }
                else{
                        printf("%d ", local[i]);
                }
        }
    printf("\n");
}
